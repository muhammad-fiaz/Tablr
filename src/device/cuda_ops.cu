#include "hip/hip_runtime.h"
/**
 * @file cuda_ops.cu
 * @brief CUDA device operations implementation
 * @author Muhammad Fiaz
 * @license Apache-2.0
 * 
 * This file provides CUDA-specific operations for GPU acceleration
 * including memory management, device queries, and kernel operations.
 */

#include "tablr/core/types.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

/**
 * @brief Get number of CUDA devices
 * @return Number of available CUDA devices
 */
int tablr_cuda_device_count(void) {
    int count = 0;
    hipError_t err = hipGetDeviceCount(&count);
    if (err != hipSuccess) {
        return 0;
    }
    return count;
}

/**
 * @brief Check if CUDA is available
 * @return true if CUDA is available, false otherwise
 */
bool tablr_cuda_available(void) {
    return tablr_cuda_device_count() > 0;
}

/**
 * @brief Synchronize CUDA device
 * @return true on success, false on failure
 */
bool tablr_cuda_synchronize(void) {
    hipError_t err = hipDeviceSynchronize();
    return err == hipSuccess;
}

/**
 * @brief Get CUDA device properties
 * @param device_id Device ID to query
 */
void tablr_cuda_device_properties(int device_id) {
    hipDeviceProp_t prop;
    hipError_t err = hipGetDeviceProperties(&prop, device_id);
    
    if (err != hipSuccess) {
        printf("Failed to get device properties\n");
        return;
    }
    
    printf("CUDA Device %d: %s\n", device_id, prop.name);
    printf("  Compute Capability: %d.%d\n", prop.major, prop.minor);
    printf("  Total Memory: %.2f GB\n", prop.totalGlobalMem / (1024.0 * 1024.0 * 1024.0));
    printf("  Multiprocessors: %d\n", prop.multiProcessorCount);
    printf("  Max Threads per Block: %d\n", prop.maxThreadsPerBlock);
    printf("  Warp Size: %d\n", prop.warpSize);
}

/**
 * @brief Allocate CUDA device memory
 * @param size Size in bytes
 * @return Pointer to device memory or NULL on failure
 */
void* tablr_cuda_malloc(size_t size) {
    void* ptr = NULL;
    hipError_t err = hipMalloc(&ptr, size);
    if (err != hipSuccess) {
        return NULL;
    }
    return ptr;
}

/**
 * @brief Free CUDA device memory
 * @param ptr Pointer to device memory
 */
void tablr_cuda_free(void* ptr) {
    if (ptr != NULL) {
        hipFree(ptr);
    }
}

/**
 * @brief Copy memory from host to device
 * @param dst Destination device pointer
 * @param src Source host pointer
 * @param size Size in bytes
 * @return true on success, false on failure
 */
bool tablr_cuda_memcpy_h2d(void* dst, const void* src, size_t size) {
    hipError_t err = hipMemcpy(dst, src, size, hipMemcpyHostToDevice);
    return err == hipSuccess;
}

/**
 * @brief Copy memory from device to host
 * @param dst Destination host pointer
 * @param src Source device pointer
 * @param size Size in bytes
 * @return true on success, false on failure
 */
bool tablr_cuda_memcpy_d2h(void* dst, const void* src, size_t size) {
    hipError_t err = hipMemcpy(dst, src, size, hipMemcpyDeviceToHost);
    return err == hipSuccess;
}

/**
 * @brief Copy memory from device to device
 * @param dst Destination device pointer
 * @param src Source device pointer
 * @param size Size in bytes
 * @return true on success, false on failure
 */
bool tablr_cuda_memcpy_d2d(void* dst, const void* src, size_t size) {
    hipError_t err = hipMemcpy(dst, src, size, hipMemcpyDeviceToDevice);
    return err == hipSuccess;
}

/**
 * @brief CUDA kernel for element-wise addition (float32)
 */
__global__ void cuda_add_float32_kernel(float* result, const float* a, const float* b, size_t size) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        result[idx] = a[idx] + b[idx];
    }
}

/**
 * @brief CUDA kernel for element-wise multiplication (float32)
 */
__global__ void cuda_mul_float32_kernel(float* result, const float* a, const float* b, size_t size) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        result[idx] = a[idx] * b[idx];
    }
}

/**
 * @brief CUDA kernel for scalar multiplication (float32)
 */
__global__ void cuda_scale_float32_kernel(float* result, const float* a, float scalar, size_t size) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        result[idx] = a[idx] * scalar;
    }
}

/**
 * @brief Launch CUDA addition kernel
 * @param result Result device pointer
 * @param a First operand device pointer
 * @param b Second operand device pointer
 * @param size Number of elements
 * @return true on success, false on failure
 */
bool tablr_cuda_add_float32(float* result, const float* a, const float* b, size_t size) {
    int threads = 256;
    int blocks = (size + threads - 1) / threads;
    
    cuda_add_float32_kernel<<<blocks, threads>>>(result, a, b, size);
    
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        return false;
    }
    
    return tablr_cuda_synchronize();
}

/**
 * @brief Launch CUDA multiplication kernel
 * @param result Result device pointer
 * @param a First operand device pointer
 * @param b Second operand device pointer
 * @param size Number of elements
 * @return true on success, false on failure
 */
bool tablr_cuda_mul_float32(float* result, const float* a, const float* b, size_t size) {
    int threads = 256;
    int blocks = (size + threads - 1) / threads;
    
    cuda_mul_float32_kernel<<<blocks, threads>>>(result, a, b, size);
    
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        return false;
    }
    
    return tablr_cuda_synchronize();
}

/**
 * @brief Launch CUDA scalar multiplication kernel
 * @param result Result device pointer
 * @param a Input device pointer
 * @param scalar Scalar value
 * @param size Number of elements
 * @return true on success, false on failure
 */
bool tablr_cuda_scale_float32(float* result, const float* a, float scalar, size_t size) {
    int threads = 256;
    int blocks = (size + threads - 1) / threads;
    
    cuda_scale_float32_kernel<<<blocks, threads>>>(result, a, scalar, size);
    
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        return false;
    }
    
    return tablr_cuda_synchronize();
}
